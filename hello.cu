#include <iostream>
#include <hip/hip_runtime.h>

__global__ void helloWorld() {
    printf("Hello, World from the GPU!\n");
}

int main() {
    // Launch the kernel with 1 block and 1 thread
    helloWorld<<<1, 1>>>();

    // Wait for GPU to finish
    hipDeviceSynchronize();

    std::cout << "Hello, World from the CPU!" << std::endl;

    return 0;
}
