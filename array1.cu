
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}

int main() {
    const int ARRAY_SIZE = 5;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

    int h_a[ARRAY_SIZE] = {1, 2, 3, 4, 5};
    int h_b[ARRAY_SIZE] = {10, 20, 30, 40, 50};
    int h_c[ARRAY_SIZE];

    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, ARRAY_BYTES);
    hipMalloc((void**)&d_b, ARRAY_BYTES);
    hipMalloc((void**)&d_c, ARRAY_BYTES);

    hipMemcpy(d_a, h_a, ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, ARRAY_BYTES, hipMemcpyHostToDevice);

    add<<<1, ARRAY_SIZE>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, ARRAY_BYTES, hipMemcpyDeviceToHost);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
